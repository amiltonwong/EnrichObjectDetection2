#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "cudaDecorrelateFeature.h"
#include "cudaDecorrelateFeature.cuh"

static bool debug = true;

    /*
        STAGE
                1. Return host Scrambled Sigma
                2. Return host 
    */

/*
    prhs[0] = gpu_Gamma     single precision
    prhs[1] = HOGTemplate   single precision
    prhs[2] = nonEmptyRows  int32
    prhs[3] = nonEmptyCols  int32
    prhs[4] = lambda        double
    prhs[5] = (optional) FEATURE_THRESHOLD
    prhs[6] = (optional) CG_TOLERANCE
    prhs[7] = (optional) CG_MAX_ITER
    prhs[8] = (optional) thread size
*/
////////////////////////////////////////////////////////////////////////////////
// Mex Entry
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    char const * const errId = "cudaDecorrelateFeature:InvalidInput";

    float FEATURE_THRESHOLD = 1.5f;
    float CG_TOLERANCE      = 0.001f;
    int CG_MAX_ITER         = 60;
    int TEMPLATE_WIDTH, TEMPLATE_HEIGHT, FEATURE_DIM;

    /* Choose a reasonably sized number of threads for the block. */
    int THREAD_PER_BLOCK_H  = 16;
    int THREAD_PER_BLOCK_W  = 8;
    int THREAD_PER_BLOCK_D  = 8;
    int THREAD_PER_BLOCK_2D_H = 32;
    int THREAD_PER_BLOCK_2D_W = 32;

    /* Initialize the MathWorks GPU API. */
    mxInitGPU();
    
    int SIGMA_OUT   = 0;
    int CG_OUT      = 1;

    int GAMMA_IN            = 0;
    int CENTERED_TEMPLATE_IN= 1;
    int TEMPLATE_HEIGHT_IN  = 2;
    int TEMPLATE_WIDTH_IN   = 3;
    int FEATURE_DIM_IN      = 4;
    int NON_EMPTY_ROW_IN    = 5;
    int NON_EMPTY_COL_IN    = 6;
    int LAMBDA_IN           = 7;
    int FEATURE_THRESHOLD_IN= 8;    /* Optional */
    int CG_TOLERANCE_IN     = 9;    /* Optional */
    int CG_MAX_ITER_IN      = 10;   /* Optional */
    int THREAD_SIZE_IN      = 11;   /* Optional */

    if ( (nrhs < 5) || (nrhs > 9) )
        mexErrMsgIdAndTxt(errId, "Wrong number of inputs");
    

    /* Gamma */
    if ( !mxIsGPUArray(prhs[GAMMA_IN]))
        mexErrMsgIdAndTxt(errId, "The Gamma must be real single precision array in GPU");
    const mxGPUArray *mxGamma = mxGPUCreateFromMxArray(prhs[GAMMA_IN]);
    if ( mxGPUGetClassID(mxGamma) != mxSINGLE_CLASS )
        mexErrMsgIdAndTxt(errId, "The Gamma must be real single precision array in GPU");
    const mwSize *mxGamma_Dim = mxGPUGetDimensions(mxGamma);
    const int GammaDim        = mxGamma_Dim[0];
    const float *d_Gamma      = (float *)mxGPUGetDataReadOnly(mxGamma);


    /* Centered Template */
    const mxArray *mxCenteredTemplate = prhs[CENTERED_TEMPLATE_IN];
    if ( mxGetClassID(mxCenteredTemplate) != mxSINGLE_CLASS )
        mexErrMsgTxt("Invalid input: hog template");
    float * h_centered_template = (float *)mxGetPr(mxCenteredTemplate);
    
    /* Template height */
    if (mxGetClassID(prhs[TEMPLATE_HEIGHT_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: template height");
    TEMPLATE_HEIGHT = (float)mxGetScalar(prhs[TEMPLATE_HEIGHT_IN]);


    /* Template height */
    if (mxGetClassID(prhs[TEMPLATE_WIDTH_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: template width");
    TEMPLATE_WIDTH = (float)mxGetScalar(prhs[TEMPLATE_WIDTH_IN]);


    /* Feature dimension */
    if (mxGetClassID(prhs[FEATURE_DIM_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: feature dimension");
    FEATURE_DIM = (float)mxGetScalar(prhs[FEATURE_DIM_IN]);


    /* Non Empty Col and Row Index */
    const mxArray *mxNonEmptyRows       = prhs[NON_EMPTY_ROW_IN];
    const mxArray *mxNonEmptyCols       = prhs[NON_EMPTY_COL_IN];
    const mwSize  *mxNonEmptyRowsDim    = mxGetDimensions(mxNonEmptyRows);
    const mwSize  *mxNonEmptyColsDim    = mxGetDimensions(mxNonEmptyCols);
    if( mxNonEmptyRowsDim[0] != mxNonEmptyColsDim[0] ||
        mxNonEmptyRowsDim[1] != mxNonEmptyColsDim[1] ||
        mxGetClassID(mxNonEmptyRows) != mxINT32_CLASS ||
        mxGetClassID(mxNonEmptyCols) != mxINT32_CLASS)
        mexErrMsgIdAndTxt(errId, "Invalid non empty indexes");
    int N_ACTIVE_CELL = max(mxNonEmptyRowsDim[0], mxNonEmptyRowsDim[1]);
    int *h_nonEmptyRows = (int*)mxGetPr(mxNonEmptyRows);
    int *h_nonEmptyCols = (int*)mxGetPr(mxNonEmptyCols);


    /* Lambda, added to the diagonals of Sigma */
    if (mxGetClassID(prhs[LAMBDA_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: lambda");
    float lambda = (float)mxGetScalar(prhs[LAMBDA_IN]);


    /* FEATURE_THRESHOLD */
    if (nrhs > FEATURE_THRESHOLD_IN && mxGetClassID(prhs[FEATURE_THRESHOLD_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: feature threshold");
    if (nrhs > FEATURE_THRESHOLD_IN)
        FEATURE_THRESHOLD = (float)mxGetScalar(prhs[FEATURE_THRESHOLD_IN]);


    /* CG_TOLERANCE */
    if (nrhs > CG_TOLERANCE_IN && mxGetClassID(prhs[CG_TOLERANCE_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: CG_TOLERANCE");
    if (nrhs > CG_TOLERANCE_IN)
        CG_TOLERANCE = (float)mxGetScalar(prhs[CG_TOLERANCE_IN]);


    /* CG_MAX_ITER */
    if (nrhs > CG_MAX_ITER_IN && mxGetClassID(prhs[CG_MAX_ITER_IN]) != mxDOUBLE_CLASS)
        mexErrMsgTxt("Invalid input: CG_MAX_ITER");
    if (nrhs > CG_MAX_ITER_IN)
        CG_MAX_ITER = (int)mxGetScalar(prhs[CG_MAX_ITER_IN]);



    /* Check the Thread Size Parameters */
    if ( nrhs > THREAD_SIZE_IN  && mxGetNumberOfElements(prhs[THREAD_SIZE_IN]) != 5)
        mexErrMsgIdAndTxt(errId, "CUDA Thread Size must be 4 integers : THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D, THREAD_PER_BLOCK_2D_H, THREAD_PER_BLOCK_2D_W\nYou must choose size such that total thread will not be larger than MaxThreadsPerBlock");

    if ( nrhs > THREAD_SIZE_IN ){
        const double* threadSize = (double *)mxGetData(prhs[THREAD_SIZE_IN]);
        THREAD_PER_BLOCK_H = (int)threadSize[0];
        THREAD_PER_BLOCK_W = (int)threadSize[1];
        THREAD_PER_BLOCK_D = (int)threadSize[2];
        THREAD_PER_BLOCK_2D_H = (int)threadSize[3];
        THREAD_PER_BLOCK_2D_W = (int)threadSize[4];
        if(debug) fprintf(stderr,"Thread size: H=%d, W=%d, D=%d, 2D=%d\n",
                                    THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D, 
                                    THREAD_PER_BLOCK_2D_H, THREAD_PER_BLOCK_2D_W);
    }


    // hipDeviceProp_t dev;
    // hipGetDeviceProperties(&dev,0);
    // int success = checkDeviceProp(dev);
    
    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);
    /* Find number of cuda capable devices */
    // CUDA_SAFE_CALL(hipGetDeviceCount(&N_GPU));
    // if(debug) fprintf(stderr, "CUDA-capable device count: %i\n", N_GPU);

    /* Setup Variables */
    int N = N_ACTIVE_CELL * FEATURE_DIM;

    /* Set block size and thread size */
    // dim3 threadBlock3D(THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D);
    // dim3 dataBlockGrid3D( iDivUp(WIDTH, threadBlock3D.x), 
    //                       iDivUp(HEIGHT, threadBlock3D.y), 
    //                       iDivUp(FEATURE_DIM, threadBlock3D.z));

    dim3 threadBlock2D( THREAD_PER_BLOCK_2D_W, THREAD_PER_BLOCK_2D_H);
    dim3 dataBlockGrid2D( iDivUp(N, threadBlock2D.x), 
                          iDivUp(N, threadBlock2D.y));


    thrust::device_vector<float> vec_d_Sigma(N * N);
    float* d_Sigma  = thrust::raw_pointer_cast(&vec_d_Sigma[0]);

    thrust::device_vector<int> vec_d_nonEmptyRows(h_nonEmptyRows, h_nonEmptyRows + N_ACTIVE_CELL);
    thrust::device_vector<int> vec_d_nonEmptyCols(h_nonEmptyCols, h_nonEmptyCols + N_ACTIVE_CELL);
    int* d_nonEmptyRows  = thrust::raw_pointer_cast(&vec_d_nonEmptyRows[0]);
    int* d_nonEmptyCols  = thrust::raw_pointer_cast(&vec_d_nonEmptyCols[0]);
    scrambleGammaToSigma<<<dataBlockGrid2D, threadBlock2D>>>( d_Sigma,
                                d_Gamma,
                                lambda,
                                d_nonEmptyRows, 
                                d_nonEmptyCols,
                                GammaDim, FEATURE_DIM, N_ACTIVE_CELL );
    

    /////////////////////////////////////////////
    ///    DEBUG
    /////////////////////////////////////////////
    if (debug){
        // thrust::host_vector<float> vec_h_Sigma = vec_d_Sigma;
        mwSize mwSigma[2];
        mwSigma[0] = N; mwSigma[1] = N;
        plhs[SIGMA_OUT] = mxCreateNumericArray(2, mwSigma, mxSINGLE_CLASS, mxREAL);
        float* h_Sigma = (float *)mxGetData(plhs[SIGMA_OUT]);
        hipMemcpy(h_Sigma, d_Sigma, N * N * sizeof(float) ,hipMemcpyDeviceToHost);
        // mxFree(mwSigma);
    }
    /////////////////////////////////////////////


    // Initial point is at the origin
    thrust::device_vector<float> vec_d_x(N, 0);
    thrust::device_vector<float> vec_d_r(h_centered_template, h_centered_template + N);
    thrust::device_vector<float> vec_d_p(N);
    thrust::device_vector<float> vec_d_Ax(N);

    float* d_x  = thrust::raw_pointer_cast(&vec_d_x[0]);
    float* d_r  = thrust::raw_pointer_cast(&vec_d_r[0]);
    float* d_p  = thrust::raw_pointer_cast(&vec_d_p[0]);
    float* d_Ax = thrust::raw_pointer_cast(&vec_d_Ax[0]);

    float alpha     = 1.0f;
    float alpham1   = -1.0f;
    float beta      = 0.0f;
    float a, b, na, r0, r1, dot;

    // Ax = A * x
    // y = α op(A) * x + β * y
    hipblasStatus_t = hipblasSgemv(cublasHandle, HIPBLAS_OP_N, 
                N, N, 
                &alpha,
                d_Sigma, N,
                d_x, 1,
                &beta, 
                d_Ax, 1);
    checkCudaErrors(hipblasStatus_t);

    // r = -A * x = - Ax
    // y = α x + y
    hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    checkCudaErrors(hipblasStatus_t);

    // r1 = r^T r
    hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
    checkCudaErrors(hipblasStatus_t);

    int k = 1;

    while (r1 > CG_TOLERANCE*CG_TOLERANCE && k <= CG_MAX_ITER)
    {
        if (k > 1)
        {
            b = r1 / r0;
            // p = bp
            hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
            checkCudaErrors(hipblasStatus_t);
            // p = r + p
            hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
            checkCudaErrors(hipblasStatus_t);
        }
        else
        {
            // Initialize p = r
            // p = r
            hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
            checkCudaErrors(hipblasStatus_t);
        }

        // Ax = A * p
        // y = α op(A) * p + β * y
        hipblasStatus_t = hipblasSgemv(cublasHandle, HIPBLAS_OP_N, 
                N, N, 
                &alpha,
                d_Sigma, N,
                d_p, 1,
                &beta, 
                d_Ax, 1);
        checkCudaErrors(hipblasStatus_t);

        // dot = p^T * Ax = p^T * A * p
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        checkCudaErrors(hipblasStatus_t);
        a = r1 / dot;

        // x = a * p + x
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        checkCudaErrors(hipblasStatus_t);
        na = -a;
        // r = - a * Ax = - a * A * p
        hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);
        checkCudaErrors(hipblasStatus_t);
        r0 = r1;
        // r1 = r^T * r
        hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        checkCudaErrors(hipblasStatus_t);

        hipDeviceSynchronize();
        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }


    mwSize mwN[1];
    mwN[0] = N;
    plhs[CG_OUT] = mxCreateNumericArray(1, mwN, mxSINGLE_CLASS, mxREAL);
    float* h_CG = (float *)mxGetData(plhs[CG_OUT]);
    hipMemcpy(h_CG, d_x, N * sizeof(float) ,hipMemcpyDeviceToHost);

    hipblasDestroy(cublasHandle);
    mxGPUDestroyGPUArray(mxGamma);
    // mxFree(mwN);

    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    // hipDeviceReset();
}
