
#include <hip/hip_runtime.h>
__global__ void scrambleGammaToSigma( float* Sigma, float* Gamma, float lambda, int* nonEmptyRows, int* nonEmptyCols, int GammaDim, int HOGDim, int nNonEmptyCells )
{
    int r = blockDim.x * blockIdx.x + threadIdx.x; // rows
    int c = blockDim.y * blockIdx.y + threadIdx.y; // cols 

    int sigmaDim = HOGDim * nNonEmptyCells;
    if( r < sigmaDim && c < sigmaDim ){
        int HOG_row_idx = r % HOGDim;
        int HOG_col_idx = c % HOGDim;

        int currCellIdx = r / HOGDim;
        int otherCellIdx = c / HOGDim;

        int gammaRowIdx = abs( nonEmptyRows[currCellIdx] - nonEmptyRows[otherCellIdx] );
        int gammaColIdx = abs( nonEmptyCols[currCellIdx] - nonEmptyCols[otherCellIdx] );
        Sigma[r + c * sigmaDim] = Gamma[ ((gammaRowIdx * HOGDim) + HOG_row_idx) + ( ( gammaColIdx * HOGDim ) + HOG_col_idx ) * GammaDim ]; // + (r==c)?lambda:0 ;
        if (r == c) Sigma[r + c * sigmaDim] += lambda;
    }
}